#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipSetDevice(0); // 选择第 0 号 GPU

    void *d_mem;
    size_t size = 7L * 1024 * 1024 * 1024;  // 2GB

    hipError_t err = hipMalloc(&d_mem, size);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    std::cout << "Allocated 2GB on GPU, press Enter to release." << std::endl;
    std::cin.get();

    hipFree(d_mem);
    return 0;
}
